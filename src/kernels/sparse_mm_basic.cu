/***********************************************************************
*  basic_spmm_tc.cu — Sparse(ish) Matrix × Matrix with optional TC    *
*                                                                    *
*  • 自动生成 A 的稀疏掩码并按掩码跳过零‑tile                         *
*  • 若 tile 大小能被 16 整除，且硬件 Compute Capability ≥ 75，      *
*    则在每个被访问的 tile 上使用 Tensor Core (WMMA + TF32)          *
*  • 其余情况退化为改进后的 shared‑memory SPMM 路径（无正确性缺陷）  *
*                                                                    *
*  编译示例（Ampere / Hopper 等支持 TF32 TensorCore 的 GPU）：        *
*      nvcc -O3 -arch=sm_80 -DBLK_X=16 -DBLK_Y=16 -DPAD=1 \          *
*           -DUSE_TENSOR_CORE basic_spmm_tc.cu -o spmm               *
***********************************************************************/

#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <cstdlib>

#ifndef BLK_X
#   define BLK_X 16      // output tile 宽度（软件路径）
#endif
#ifndef BLK_Y
#   define BLK_Y 16      // output tile 高度（软件路径）
#endif
#ifndef PAD
#   define PAD   1       // 共享内存行末 padding → 避免 bank conflict
#endif

/* ===================== 工具宏 & util ====================== */
#define CHECK_CUDA(call)                                                      \
do {                                                                          \
    hipError_t _e = (call);                                                  \
    if (_e != hipSuccess) {                                                  \
        fprintf(stderr,"CUDA error %s:%d : %s\n",                             \
                __FILE__,__LINE__, hipGetErrorString(_e));                   \
        exit(EXIT_FAILURE);                                                   \
    }                                                                         \
} while (0)

static inline double to_ms(hipEvent_t beg, hipEvent_t end)
{
    float ms = 0.f;
    hipEventElapsedTime(&ms, beg, end);
    return static_cast<double>(ms);
}

/* ========================================================== *
 *                     Kernel 1 : buildMask                   *
 * ========================================================== */
__global__ void buildMask(const float *__restrict__ A,
                          uint8_t     *__restrict__ mask,
                          int K, int N, int tile)
{
    const int nt = blockIdx.x;
    const int kt = blockIdx.y;
    const int nTiles = gridDim.x;

    const int k0   = kt * tile;
    const int n0   = nt * tile;
    const int kEnd = min(k0 + tile, K);
    const int nEnd = min(n0 + tile, N);

    bool hasNZ = false;
    for (int k = k0 + threadIdx.x; k < kEnd && !hasNZ; k += BLK_X)
        for (int n = n0 + threadIdx.y; n < nEnd && !hasNZ; n += BLK_Y)
            if (__ldg(&A[k + (size_t)n * K]) != 0.f) { hasNZ = true; break; }

    __syncthreads();
    hasNZ = __syncthreads_or(hasNZ);
    if (threadIdx.x == 0 && threadIdx.y == 0)
        mask[kt * nTiles + nt] = static_cast<uint8_t>(hasNZ);
}

/* ========================================================== *
 *              Kernel 2 : shared‑mem SPMM (改进版)            *
 * ========================================================== */
__launch_bounds__(BLK_X * BLK_Y, 2)
__global__ void spmmTile_sw(const float *__restrict__ W,
                            const float *__restrict__ A,
                            const float *__restrict__ B,
                                  float *__restrict__ P,
                            const uint8_t *__restrict__ mask,
                            int M, int K, int N, int tile)
{
    /* 本线程处理的全局 (m,n) */
    const int m = blockIdx.y * BLK_Y + threadIdx.y;
    const int n = blockIdx.x * BLK_X + threadIdx.x;
    const bool valid_m = (m < M);
    const bool valid_n = (n < N);

    const int nTiles  = (N + tile - 1) / tile;
    const int kTiles  = (K + tile - 1) / tile;
    const int nTileId = n / tile;

    /* ----------- 动态共享内存布局 ----------- *
     *  shW : tile × BLK_Y      (行步长 = BLK_Y+PAD)
     *  shA : tile × tile       (行步长 = tile+PAD)
     */
    extern __shared__ float sh[];
    const int strideW = BLK_Y + PAD;
    const int strideA = tile   + PAD;
    float *shW = sh;
    float *shA = sh + tile * strideW;

    float acc = 0.f;

    for (int kt = 0; kt < kTiles; ++kt)
    {
        if (!mask[kt * nTiles + nTileId])      // 整个 tile 全零
            continue;

        const int kBase = kt * tile;
        const int rows  = min(tile, K - kBase);

        /* ---- 1. W 子块读入共享内存 (col‑major → row in shmem) ---- */
        for (int kk = threadIdx.x; kk < rows; kk += BLK_X)
        {
            const int kIdx = kBase + kk;
            float w = 0.f;
            if (valid_m)  w = __ldg(&W[m + (size_t)kIdx * M]);
            shW[kk * strideW + threadIdx.y] = w;
        }

        /* ---- 2. 清零 A‑tile 有效区 ---- */
        for (int idx = threadIdx.y * BLK_X + threadIdx.x;
             idx < rows * strideA;
             idx += BLK_X * BLK_Y)
            shA[idx] = 0.f;
        __syncthreads();

        /* ---- 3. 把 A 子块搬进共享内存 (col‑major → row‑major in shmem) ---- */
        const int n0       = nTileId * tile;
        const int colsIn   = min(tile, N - n0);
        for (int kk = threadIdx.y; kk < rows; kk += BLK_Y)
            for (int nn = threadIdx.x; nn < colsIn; nn += BLK_X)
            {
                size_t gOff = (kBase + kk) + (size_t)(n0 + nn) * K;
                shA[kk * strideA + nn] = __ldg(&A[gOff]);
            }
        __syncthreads();

        /* ---- 4. 本线程完成 rows × colsIn 的乘加 ---- */
        if (valid_m && valid_n)
        {
            const int local_n = n - n0;
#pragma unroll 4
            for (int kk = 0; kk < rows; ++kk)
            {
                float a = (local_n < colsIn) ?
                          shA[kk * strideA + local_n] : 0.f;
                acc += shW[kk * strideW + threadIdx.y] * a;
            }
        }
        __syncthreads();
    }

    /* ---- 5. 写回 ---- */
    if (valid_m && valid_n)
    {
        const size_t off = m + (size_t)n * M;
        P[off] = acc + __ldg(&B[off]);
    }
}

/* ========================================================== *
 *       (可选) Kernel 3 : Tensor Core (WMMA‑TF32) 路径         *
 * ========================================================== */
#if defined(USE_TENSOR_CORE) && (__CUDA_ARCH__ >= 750)
#include <mma.h>
using namespace nvcuda;

__launch_bounds__(32, 4)      // 一个 warp 即一个 16×16 输出 tile
__global__ void spmmTile_tc(const float *__restrict__ W,
                            const float *__restrict__ A,
                            const float *__restrict__ B,
                                  float *__restrict__ P,
                            const uint8_t *__restrict__ mask,
                            int M, int K, int N, int tile)   // tile 必须是 16 的倍数
{
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    const int mTile = blockIdx.y;                 // 16 行块
    const int nTile = blockIdx.x;                 // 16 列块

    const int row_base = mTile * WMMA_M;
    const int col_base = nTile * WMMA_N;

    const int nTiles   = (N + tile - 1) / tile;
    const int kTiles   = (K + tile - 1) / tile;
    const int nTileId  = col_base / tile;         // 对应 buildMask 的列‑tile 号

    /* 累加器 fragment 初始化为 0 */
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);

    /* 每次循环处理 K 方向一个 “tile”(用户指定大小, 多为 16/32/64) */
    for (int kt = 0; kt < kTiles; ++kt)
    {
        if (!mask[kt * nTiles + nTileId])         // 整 tile 为 0 → 跳过
            continue;

        const int kBase  = kt * tile;
        const int rowsK  = min(tile, K - kBase);

        /* tile 里按 16 分块做 mma */
        for (int ks = 0; ks < rowsK; ks += WMMA_K)
        {
            const int kSub = kBase + ks;
            /* 共有 rowsK /16 (向上取整) 次子 mma */

            /* ===================================================== *
             *          1. 装入 W / A 到共享内存 (row‑major)           *
             * ===================================================== */
            __shared__ float shW[WMMA_M * WMMA_K];   // 256B
            __shared__ float shA[WMMA_K * WMMA_N];   // 256B

            const int lane = threadIdx.x;            // 0..31

            /* 让 32 线程一次性 copy 两个 16×16 字板块 */
            for (int i = lane; i < WMMA_M * WMMA_K; i += 32)
            {
                int r = i / WMMA_K;
                int c = i % WMMA_K;
                int gR = row_base + r;
                int gC = kSub     + c;
                shW[i] = (gR < M && gC < K) ? __ldg(&W[gR + (size_t)gC * M]) : 0.0f;
            }
            for (int i = lane; i < WMMA_K * WMMA_N; i += 32)
            {
                int r = i / WMMA_N;
                int c = i % WMMA_N;
                int gR = kSub     + r;
                int gC = col_base + c;
                shA[i] = (gR < K && gC < N) ? __ldg(&A[gR + (size_t)gC * K]) : 0.0f;
            }
            __syncthreads();

            /* ===================================================== *
             *               2. WMMA 计算 (row_major)                 *
             * ===================================================== */
            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,
                           wmma::precision::tf32, wmma::row_major> a_frag;
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K,
                           wmma::precision::tf32, wmma::row_major> b_frag;

            wmma::load_matrix_sync(a_frag, shW, WMMA_K);
            wmma::load_matrix_sync(b_frag, shA, WMMA_N);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

            __syncthreads();
        } /* --- end ks --- */
    }     /* --- end kt --- */

    /* ===================================================== *
     *             3. store 结果到全局内存                   *
     * ===================================================== */
    __shared__ float shC[WMMA_M * WMMA_N];   // 256B
    wmma::store_matrix_sync(shC, c_frag, WMMA_N, wmma::mem_row_major);
    __syncthreads();

    const int lane = threadIdx.x;
    for (int i = lane; i < WMMA_M * WMMA_N; i += 32)
    {
        int r = i / WMMA_N;
        int c = i % WMMA_N;
        int gR = row_base + r;
        int gC = col_base + c;
        if (gR < M && gC < N)
        {
            size_t off = gR + (size_t)gC * M;
            P[off] = shC[i] + __ldg(&B[off]);
        }
    }
}
#endif  /* USE_TENSOR_CORE && arch>=75 */

/* ========================================================== *
 *                  Host wrapper : runBasicSparse             *
 * ========================================================== */
double runBasicSparse(const float *dW, const float *dA,
                      const float *dB,       float *dP,
                      int M, int K, int N, int tile)
{
    const int nTiles = (N + tile - 1) / tile;
    const int kTiles = (K + tile - 1) / tile;

    /* ---- 0. 生成稀疏掩码 ---- */
    uint8_t *dMask = nullptr;
    CHECK_CUDA(hipMalloc(&dMask, nTiles * kTiles * sizeof(uint8_t)));

    dim3 thrMask(BLK_X, BLK_Y);
    dim3 gridMask(nTiles, kTiles);
    buildMask<<<gridMask, thrMask>>>(dA, dMask, K, N, tile);
    CHECK_CUDA(hipGetLastError());

    /* ---- 1. 决定使用哪条计算路径 ---- */
    bool useTC =
#if defined(USE_TENSOR_CORE)
        (tile % 16 == 0) && (cudaDeviceProp{}.major >= 8 /* conservative */);
#else
        false;
#endif

    hipEvent_t t0, t1;
    hipEventCreate(&t0);
    hipEventCreate(&t1);
    hipEventRecord(t0);

    if (useTC)
    {
#if defined(USE_TENSOR_CORE)
        dim3 grid((N + 15) / 16, (M + 15) / 16);
        spmmTile_tc<<<grid, 32>>>(dW, dA, dB, dP,
                                  dMask, M, K, N, tile);
        CHECK_CUDA(cudaGetLastError());
#else
        (void)0;   // 不会走到
#endif
    }
    else
    {
        dim3 thr(BLK_X, BLK_Y);
        dim3 grid((N + BLK_X - 1) / BLK_X,
                  (M + BLK_Y - 1) / BLK_Y);

        size_t shBytes = (size_t)tile * (BLK_Y + PAD) +
                         (size_t)tile * (tile   + PAD);
        shBytes *= sizeof(float);

        spmmTile_sw<<<grid, thr, shBytes>>>(dW, dA, dB, dP,
                                            dMask, M, K, N, tile);
        CHECK_CUDA(hipGetLastError());
    }

    hipEventRecord(t1);
    CHECK_CUDA(hipEventSynchronize(t1));

    double ms = to_ms(t0, t1);
    hipFree(dMask);
    hipEventDestroy(t0);
    hipEventDestroy(t1);
    return ms;
}
