#include "../../include/benchmark.h"

// Standard cuBLAS SGEMM baseline
double denseSGEMM(hipblasHandle_t handle, const float* dW, const float* dA,
                 const float* dB, float* dP, int M, int K, int N)
{
    CHECK_CUDA(hipMemcpy(dP, dB, (size_t)M * N * sizeof(float),
                          hipMemcpyDeviceToDevice));
    float alpha = 1.f, beta = 1.f;
    hipEvent_t s, e; hipEventCreate(&s); hipEventCreate(&e);
    hipEventRecord(s);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
                dW, M, dA, K, &beta, dP, M);
    hipEventRecord(e); hipEventSynchronize(e);
    double ms = to_ms(s, e);
    hipEventDestroy(s); hipEventDestroy(e);
    return ms;
}

// cuBLASLt with TF32
double denseLtTF32(hipblasLtHandle_t lt, const float* dW, const float* dA,
                  const float* dB, float* dP, int M, int K, int N)
{
    hipblasLtMatmulDesc_t op; hipblasLtMatrixLayout_t a, b, c;
    hipblasLtMatmulDescCreate(&op, HIPBLAS_COMPUTE_32F_FAST_TF32, HIP_R_32F);
    hipblasLtMatrixLayoutCreate(&a, HIP_R_32F, K, N, K);
    hipblasLtMatrixLayoutCreate(&b, HIP_R_32F, M, K, M);
    hipblasLtMatrixLayoutCreate(&c, HIP_R_32F, M, N, M);

    CHECK_CUDA(hipMemcpy(dP, dB, (size_t)M * N * sizeof(float),
                          hipMemcpyDeviceToDevice));
    float alpha = 1.f, beta = 1.f;
    hipEvent_t s, e; hipEventCreate(&s); hipEventCreate(&e);
    hipEventRecord(s);
    hipblasLtMatmul(lt, op, &alpha, dW, b, dA, a, &beta, dP, c, dP, c,
                   nullptr, nullptr, 0, 0);
    hipEventRecord(e); hipEventSynchronize(e);
    double ms = to_ms(s, e);
    hipEventDestroy(s); hipEventDestroy(e);
    hipblasLtMatrixLayoutDestroy(a);
    hipblasLtMatrixLayoutDestroy(b);
    hipblasLtMatrixLayoutDestroy(c);
    hipblasLtMatmulDescDestroy(op);
    return ms;
}

// cuBLASLt with heuristic algorithm selection
double denseLtOptimal(hipblasLtHandle_t lt, const float* dW, const float* dA,
                     const float* dB, float* dP, int M, int K, int N)
{
    hipblasLtMatmulDesc_t op; hipblasLtMatrixLayout_t a, b, c;
    hipblasLtMatmulDescCreate(&op, HIPBLAS_COMPUTE_32F_FAST_TF32, HIP_R_32F);
    hipblasLtMatrixLayoutCreate(&a, HIP_R_32F, K, N, K);
    hipblasLtMatrixLayoutCreate(&b, HIP_R_32F, M, K, M);
    hipblasLtMatrixLayoutCreate(&c, HIP_R_32F, M, N, M);

    // Find best algorithm using heuristics
    hipblasLtMatmulPreference_t pref;
    hipblasLtMatmulPreferenceCreate(&pref);
    size_t workspaceSize = 1024 * 1024 * 32; // 32MB workspace
    hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                         &workspaceSize, sizeof(workspaceSize));
    
    hipblasLtMatmulHeuristicResult_t heuristic;
    int returnedAlgoCount;
    hipblasLtMatmulAlgoGetHeuristic(lt, op, a, b, c, c, pref, 1, &heuristic, &returnedAlgoCount);
    
    void* workspace = nullptr;
    if(returnedAlgoCount > 0) {
        hipMalloc(&workspace, workspaceSize);
    }

    CHECK_CUDA(hipMemcpy(dP, dB, (size_t)M * N * sizeof(float),
                          hipMemcpyDeviceToDevice));
    float alpha = 1.f, beta = 1.f;
    hipEvent_t s, e; hipEventCreate(&s); hipEventCreate(&e);
    hipEventRecord(s);
    
    if(returnedAlgoCount > 0) {
        hipblasLtMatmul(lt, op, &alpha, dW, b, dA, a, &beta, dP, c, dP, c,
                       &heuristic.algo, workspace, workspaceSize, 0);
    } else {
        hipblasLtMatmul(lt, op, &alpha, dW, b, dA, a, &beta, dP, c, dP, c,
                       nullptr, nullptr, 0, 0);
    }
    
    hipEventRecord(e); hipEventSynchronize(e);
    double ms = to_ms(s, e);
    hipEventDestroy(s); hipEventDestroy(e);
    
    if(workspace) hipFree(workspace);
    hipblasLtMatmulPreferenceDestroy(pref);
    hipblasLtMatrixLayoutDestroy(a);
    hipblasLtMatrixLayoutDestroy(b);
    hipblasLtMatrixLayoutDestroy(c);
    hipblasLtMatmulDescDestroy(op);
    return ms;
}