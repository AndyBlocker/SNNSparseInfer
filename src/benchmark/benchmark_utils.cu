#include "hip/hip_runtime.h"
#include "../../include/benchmark.h"
#include "../../include/kernels.h"

double calculateRMSError(const std::vector<float>& ref,
                        const std::vector<float>& test)
{
    double diff2 = 0, ref2 = 0;
    for(size_t i = 0; i < ref.size(); ++i) {
        const double d = ref[i] - test[i];
        diff2 += d * d; 
        ref2 += ref[i] * ref[i];
    }
    return std::sqrt(diff2 / ref2);
}

BenchmarkResult runCompleteBenchmark(
    const float* dW, const float* dA, const float* dB, float* dP,
    int M, int K, int N, int tile)
{
    BenchmarkResult result = {};
    
    // Create cuBLAS handles
    hipblasHandle_t handle; hipblasCreate(&handle);
    hipblasLtHandle_t lt; hipblasLtCreate(&lt);
    
    size_t szB = (size_t)M * N;
    std::vector<float> hPd(szB), hPs_basic(szB), hPs_pipeline(szB), hPs_warp_gather(szB);
    
    // Warmup runs
    denseSGEMM(handle, dW, dA, dB, dP, M, K, N);
    denseLtTF32(lt, dW, dA, dB, dP, M, K, N);
    denseLtOptimal(lt, dW, dA, dB, dP, M, K, N);
    
    // Dense baselines
    result.ms_sgemm = denseSGEMM(handle, dW, dA, dB, dP, M, K, N);
    CHECK_CUDA(hipMemcpy(hPd.data(), dP, szB * sizeof(float),
                          hipMemcpyDeviceToHost));
    
    result.ms_lt_tf32 = denseLtTF32(lt, dW, dA, dB, dP, M, K, N);
    result.ms_lt_optimal = denseLtOptimal(lt, dW, dA, dB, dP, M, K, N);
    
    result.best_dense_ms = std::min({result.ms_sgemm, result.ms_lt_tf32, result.ms_lt_optimal});
    
    // Sparse implementations
    result.ms_sparse_basic = runBasicSparse(dW, dA, dB, dP, M, K, N, tile);
    CHECK_CUDA(hipMemcpy(hPs_basic.data(), dP, szB * sizeof(float),
                          hipMemcpyDeviceToHost));
    
    result.ms_sparse_pipeline = runPipelineSparse(dW, dA, dB, dP, M, K, N, tile);
    CHECK_CUDA(hipMemcpy(hPs_pipeline.data(), dP, szB * sizeof(float),
                          hipMemcpyDeviceToHost));
    
    result.ms_sparse_warp_gather = runWarpGatherSparse(dW, dA, dB, dP, M, K, N, tile);
    CHECK_CUDA(hipMemcpy(hPs_warp_gather.data(), dP, szB * sizeof(float),
                          hipMemcpyDeviceToHost));
    
    // Calculate errors
    result.rms_error_basic = calculateRMSError(hPd, hPs_basic);
    result.rms_error_pipeline = calculateRMSError(hPd, hPs_pipeline);
    result.rms_error_warp_gather = calculateRMSError(hPd, hPs_warp_gather);
    
    // Cleanup
    hipblasDestroy(handle);
    hipblasLtDestroy(lt);
    
    return result;
}